#include "hip/hip_runtime.h"
fn range(start, stop=null, stride=null) {
	real_stride = if (stride == null) 1 else stride
	real_stop = if (stop == null) start else stop
	real_start = if (stop == null) 0 else start
	\if (real_start < real_stop) {
		val = real_start
		real_start += real_stride
		val
	}
}

fn int(chrs) {
	rv = 0
	for (chr in chrs) {
		ch = chr - '0' 
		rv = 10 * rv + ch
	}
	rv
}

fn map(l, f) {
	rv = []
	for (i in l) {
		rv += [f(i)]
	}
	rv
}

fn reduce(list, fun)  {
	rv = list[0]
	for (i in range(1, len(list))) {
		rv = fun(rv, list[i])
	}
	rv
}

fn fold(list, fun, val) {
	for (v in list) {
		val = fun(val, v)
	}
	val
}

fn filter(list, predicate) {
	rv = []
	for (val in list) {
		if (predicate(val)) {
			rv += [val]
		}
	}
	rv
}

fn sum(l) reduce(l, \0 + \1)

fn sort(list) {
	fn swap(l, a, b) {
		t = l[a]
		l[a] = l[b]
		l[b] = t
	}

	fn partition(list, low, high) {
		pivot = list[high]
		i = low - 1
		j = low
		while(j < high) {
			if(list[j] < pivot) {
				i++
				swap(list, i, j)
			}
			j++
		}
		swap(list, i+1, high)
		i + 1
	}

	fn qsort(list, low, high) {
		if (low < high) {
			pi = partition(list, low, high)
			qsort(list, low, pi - 1)
			qsort(list, pi + 1, high)
		}
	}
	qsort(list, 0, len(list) -1)
}

fn abs(a) if (a < 0) -a else a
fn min(a, b) if (a < b) a else b
fn max(a, b) if (a > b) a else b

fn str_split(input, sep) {
	fn streq(str, offset, haystack) {
		i = 0
		rv = true
		for (i in range(len(haystack))) {
			if (i + offset >= len(str) || haystack[i] != str[i + offset]) {
				rv = false
			}
		}
		rv
	}
	rv = []
	cur = []
	i = 0
	while(i < len(input)) {
		if (!streq(input, i, sep)) {
			cur += [input[i]]
			i++
		}
		else {
			rv += [cur]
			cur = []
			i += len(sep)
		}
	}
	rv += [cur]
	rv
}
